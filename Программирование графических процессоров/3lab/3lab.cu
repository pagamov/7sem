
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <string>
#include <string.h>
#include <math.h>

using namespace std;

#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}																\
} while(0)

typedef struct { unsigned x, y, z; } Uint4; 									// +4 294 967 295 ~ 255 * 4100 * 4100
typedef struct { int w, h, n; } info;											// pic 4100 by 4100 of one claster max pixel
																				// mb need to be changed to higher
__constant__ uchar4 cl[32];														// contant classes no more than 32 by default
__constant__ info inf[1];														// some else param that never changes
// #define SIZE_OF_PIC sizeof(uchar4) * w * h
// #define SIZE_OF_CLU sizeof(uchar4) * n

__global__ void Kmean(uchar4 * pic) {
    for (int y = blockDim.y * blockIdx.y + threadIdx.y; y < inf[0].h; y += blockDim.y * gridDim.y) {
        for (int x = blockDim.x * blockIdx.x + threadIdx.x; x < inf[0].w; x += blockDim.x * gridDim.x) {
            int resClas = -1;
            float maxDist = sqrt((float)3*(255*255))+1.0F;
			uchar4 piv = pic[x + inf[0].w * y];
            for (int i = 0; i < inf[0].n; i++) {
                float pivDist = sqrt( 					   						      \
					(((float)piv.x-(float)cl[i].x) * ((float)piv.x-(float)cl[i].x)) + \
					(((float)piv.y-(float)cl[i].y) * ((float)piv.y-(float)cl[i].y)) + \
					(((float)piv.z-(float)cl[i].z) * ((float)piv.z-(float)cl[i].z))   \
				);
                if (pivDist < maxDist) {
                    resClas = i;
                    maxDist = pivDist;
                }
            }
            pic[x + y * inf[0].w].w = (unsigned char)resClas;
        }
    }
}

__global__ void reBuild(uchar4 * pic, Uint4 * newcl) {
	int idx = threadIdx.x;
	if (idx < inf[0].n) {
		unsigned char uidx = (unsigned char)idx;
		int num = 0;
		newcl[uidx].x = 0; newcl[uidx].y = 0; newcl[uidx].z = 0;
	    for (int y = 0; y < inf[0].h; y++) {
	        for (int x = 0; x < inf[0].w; x++) {
				if (pic[x + inf[0].w * y].w == uidx) {
					newcl[uidx].x += pic[x + inf[0].w * y].x;
					newcl[uidx].y += pic[x + inf[0].w * y].y;
					newcl[uidx].z += pic[x + inf[0].w * y].z;
					num += 1;
				}
	        }
	    }
		// # if __CUDA_ARCH__>=200
	    // 	printf("%d: %d\n", idx, num);
		// #endif 
		if (num != 0) {
			newcl[uidx].x /= num;
			newcl[uidx].y /= num;
			newcl[uidx].z /= num;
		}
	}
}

int main() {
    string filename1, filename2;
    int w, h, n, x, y, flag = 1;
    cin >> filename1 >> filename2 >> n;
	
	FILE * f = fopen(filename1.c_str(), "rb");									// read data section
	fread(&w, sizeof(int), 1, f);
	fread(&h, sizeof(int), 1, f);
	uchar4 * data = (uchar4 *)malloc(sizeof(uchar4) * w * h); 					// malloc data
	fread(data, sizeof(uchar4), w * h, f);
	fclose(f);
	
	uchar4 * dev_pic;															// make dev struct for kernel
	CSC(hipMalloc(&dev_pic, sizeof(uchar4) * w * h));							// cuda malloc dev_pic
	CSC(hipMemcpy(dev_pic, data, sizeof(uchar4) * w * h, hipMemcpyHostToDevice));
	
	info infHost[1];															// infHost
	infHost[0].w = w; infHost[0].h = h; infHost[0].n = n;						// copy info data and never touch again
	CSC(hipMemcpyToSymbol(HIP_SYMBOL(inf), infHost, sizeof(info), 0, hipMemcpyHostToDevice));
	
	uchar4 clHost[32];															// read data for classes
    for (int i = 0; i < n; i++) {
		cin >> x >> y;
		clHost[i] = data[x + w * y];
	} 																			// we ll touch it every cicle
	CSC(hipMemcpyToSymbol(HIP_SYMBOL(cl), clHost, sizeof(uchar4) * 32, 0, hipMemcpyHostToDevice));
	
	Uint4 * dev_clnew;
	CSC(hipMalloc(&dev_clnew, sizeof(Uint4) * 32));							// cuda malloc dev_clnew
	
	Uint4 * clnew; 																// but why this way?
	clnew = (Uint4 *)malloc(sizeof(Uint4) * 32);								// malloc clnew
	
	while (flag) {
		Kmean <<<dim3(16, 16), dim3(32, 32)>>> (dev_pic);						// find new clasters
		// cudaDeviceSynchronize();
	
		reBuild <<<1, 32>>> (dev_pic, dev_clnew);								// save in clnew new centers by rgb
		// cudaDeviceSynchronize();
	
		CSC(hipMemcpy(clnew, dev_clnew, sizeof(Uint4) * 32, hipMemcpyDeviceToHost));
	
		// cout << "\ncount: " << count << endl;
		// cout << "clHost:" << endl;
		// for (int i = 0; i < n; i++) {
		// 	cout << (int)clHost[i].x << " " << (int)clHost[i].y << " " << (int)clHost[i].z << endl;
		// }
		// cout << "clnew:" << endl;
		// for (int i = 0; i < n; i++) {
		// 	cout << clnew[i].x << " " << clnew[i].y << " " << clnew[i].z << endl;
		// }
	
		flag = 0;
		for (int i = 0; i < n; i++) {
			if (!(clHost[i].x == (unsigned char)clnew[i].x && \
				  clHost[i].y == (unsigned char)clnew[i].y && \
				  clHost[i].z == (unsigned char)clnew[i].z)) {
					  flag = 1;
			}
		}
	
		for (int i = 0; i < n; i++) {
			clHost[i].x = (unsigned char)clnew[i].x;
			clHost[i].y = (unsigned char)clnew[i].y;
			clHost[i].z = (unsigned char)clnew[i].z;
		}
		// cout << 'here' << endl;
	
		CSC(hipMemcpyToSymbol(HIP_SYMBOL(cl), clHost, sizeof(clHost), 0, hipMemcpyHostToDevice));
	}
	
	// Kmean <<<dim3(16, 16), dim3(32, 32)>>> (dev_pic);
    
	CSC(hipGetLastError());
	
	CSC(hipMemcpy(data, dev_pic, sizeof(uchar4) * w * h, hipMemcpyDeviceToHost));
	
	CSC(hipFree(dev_pic));														// cuda free dev_pic
	CSC(hipFree(dev_clnew));													// cuda free dev_clnew

	
	f = fopen(filename2.c_str(), "wb");
	fwrite(&w, sizeof(int), 1, f);
	fwrite(&h, sizeof(int), 1, f);
	fwrite(data, sizeof(uchar4), w * h, f);
	fclose(f);

	free(clnew);																// free clnew
	
	
	free(data);																	// free data
	return 0;
}
