#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <string>
#include <string.h>
#include <math.h>

using namespace std;

#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}																\
} while(0)

// texture<uchar4, 2, hipReadModeElementType> tex;

__global__ void Kmean(uchar4 * pic, uchar4 * cl, int w, int h, int n) {
    for (int y = blockDim.y * blockIdx.y + threadIdx.y; y < h; y += blockDim.y * gridDim.y) {
        for (int x = blockDim.x * blockIdx.x + threadIdx.x; x < w; x += blockDim.x * gridDim.x) {
            int resClas = -1;
            float maxDist = 10000;
			uchar4 piv = pic[x + w * y];
            for (int i = 0; i < n; i++) {
                float pivDist = sqrt( 					   \
					pow((float)piv.x - (float)cl[i].x,2) + \
					pow((float)piv.y - (float)cl[i].y,2) + \
					pow((float)piv.z - (float)cl[i].z,2)   \
				);
                if (pivDist < maxDist) {
                    resClas = clas;
                    maxDist = pivDist;
                }
            }
            pic[x + y * im.x].w = (unsigned char)resClas;
        }
    }
}

#define SIZE_OF_PIC sizeof(uchar4) * w * h);
#define SIZE_OF_CL sizeof(int) * n * 2);
#define SIZE_OF_CLU sizeof(uchar4) * n);

int main() {
    string filename1, filename2;
    int w, h, n, x, y;
    cin >> filename1 >> filename2 >> n;
	
	// read data section
	FILE * f = fopen(filename1.c_str(), "rb");
	fread(&w, sizeof(int), 1, f);
	fread(&h, sizeof(int), 1, f);
	uchar4 * data = (uchar4 *)malloc(SIZE_OF_PIC); //#
	fread(data, sizeof(uchar4), w * h, f);
	fclose(f);
	
	// read data for classes
	uchar4 * clu = (uchar4 *)malloc(SIZE_OF_CLU); //#
    // int * cl = (int *)malloc(SIZE_OF_CL); //#
    for (int i = 0; i < n; i++) {
		cin >> x >> y;
		memcpy(clu[i], data[x + w * y], sizeof(uchar4)); // check if works
	}
	
	// delete after check if works
	for (int i = 0; i < n; i++) {
		cout << clu[i].x << " " << clu[i].y << " " << clu[i].z << " " << clu[i].w << endl;
	}
		
	// make dev struct for kernel
	hipMalloc(&dev_pic, SIZE_OF_PIC);
    hipMalloc(&dev_cl, SIZE_OF_CLU);
	// copy data to dev struct
	hipMemcpy(dev_pic, data, SIZE_OF_PIC), hipMemcpyHostToDevice);
    hipMemcpy(dev_cl, clu, SIZE_OF_CLU, hipMemcpyHostToDevice);
    
	Kmean <<<dim3(16, 16), dim3(32, 32)>>> (dev_pic, dev_cl, w, h, n);

	CSC(hipGetLastError());
	CSC(hipMemcpy(data, dev_pic, SIZE_OF_PIC, hipMemcpyDeviceToHost));
	
	CSC(hipFree(dev_pic));
    CSC(hipFree(dev_cl));
    
	f = fopen(filename2.c_str(), "wb");
	fwrite(&w, sizeof(int), 1, f);
	fwrite(&h, sizeof(int), 1, f);
	fwrite(data, sizeof(uchar4), w * h, f);
	fclose(f);

	free(data);
	free(clu);
	// free(cl);
	return 0;
}
