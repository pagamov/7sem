#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/count.h>

using namespace std;

#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}																\
} while(0)

struct comparator {												
	__host__ __device__ bool operator()(double a, double b) {
		return abs(a) < abs(b);
	}
};

__global__ void LUP_swap(double * A, int i, int n, int newidx) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int shift = blockDim.x * gridDim.x;
	double piv;
	for (int var = idx; var < n; var += shift) {
		piv = A[newidx + n * var];
        A[newidx + n * var] = A[i + n * var];
        A[i + n * var] = piv;
	}
}

__global__ void LUP_N(double * A, int i, int n) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int shift = blockDim.x * gridDim.x;
	for (int var = idx + i + 1; var < n; var += shift) {
		A[var + n * i] /= A[i + n * i];
	}
}

__global__ void LUP(double * A, int i, int n) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;
	
	int shiftx = blockDim.x * gridDim.x;
	int shifty = blockDim.y * gridDim.y;
	
	for (int var = idx + i + 1; var < n; var += shiftx) {
		for (int k = idy + i + 1; k < n; k += shifty) {
			A[var + n * k] -= A[var + n * i] * A[i + n * k];
		}
	}
}

int main() {
    int n;
	comparator comp;
    cin >> n;
    double * A = (double *)malloc(sizeof(double) * n * n);
    
    for (int y = 0; y < n; y++)
        for (int x = 0; x < n; x++)
            cin >> A[x * n + y];
            
    double * A_DEV;
    CSC(hipMalloc(&A_DEV, sizeof(double) * n * n));
    CSC(hipMemcpy(A_DEV, A, sizeof(double) * n * n, hipMemcpyHostToDevice));
    
    int * SWP = (int *)malloc(sizeof(int) * n);
    int * SWP_DEV;
    CSC(hipMalloc(&SWP_DEV, sizeof(int) * n));
    
	int newidx;
	
	int * newidxarr = (int *)malloc(sizeof(int) * n);
	
    for(int i = 0; i < n; i++) {
		thrust::device_ptr<double> d_ptr = thrust::device_pointer_cast(A_DEV) + (i * n + i);
		thrust::device_ptr<double> max = thrust::max_element(d_ptr, d_ptr + (n - i), comp);
		newidx = max - d_ptr + i;
		newidxarr[i] = newidx;
		LUP_swap <<<32,32>>> (A_DEV, i, n, newidx);
		LUP_N <<<32,32>>> (A_DEV, i, n);
		LUP <<<dim3(32,32),dim3(32,32)>>> (A_DEV, i, n);
    }
	                      
    CSC(hipMemcpy(A, A_DEV, sizeof(double) * n * n, hipMemcpyDeviceToHost));
    CSC(hipMemcpy(SWP, SWP_DEV, sizeof(int) * n, hipMemcpyDeviceToHost));
    
    for (int y = 0; y < n; y++) {
        for (int x = 0; x < n; x++)
            printf("%.10lf ", A[x * n + y]);
        printf("\n");
    }
    for (int i = 0; i < n; i++) {
		printf("%d ", newidxarr[i]);
	}
    printf("\n");
	
	free(newidxarr);
    CSC(hipFree(A_DEV));
    CSC(hipFree(SWP_DEV));
    free(A);
    free(SWP);
    return 0;
}