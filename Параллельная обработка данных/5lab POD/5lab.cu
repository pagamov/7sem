#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <thrust/swap.h>
#include <thrust/extrema.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

using namespace std;

#define BLOCK_SIZE 512
#define NUM_BLOCKS (2048+2048)

#define CSC(call)                                                   \
do {                                                                \
    hipError_t res = call;                                         \
    if (res != hipSuccess) {                                       \
        fprintf(stderr, "ERROR in %s:%d. Message: %s\n",            \
                __FILE__, __LINE__, hipGetErrorString(res));       \
        exit(0);                                                    \
    }                                                               \
} while(0)

__global__ void oddEvenSortingStep(int * A, int i, int n, int batch) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int shift = blockDim.x * gridDim.x;
    int piv;
    for (int start = idx * batch; start < n; start += shift * batch) {
        for (int j = start + (i % 2); j + 1 < min(start + batch, n); j += 2) {
            if (A[j] > A[j + 1]) {
                piv = A[j];
                A[j] = A[j + 1];
                A[j + 1] = piv;
            }
        }
    }
}

__global__ void mergeGPU(int * arr, int upd_n, int batch, int start) {
    __shared__ int l[BLOCK_SIZE];
    __shared__ int r[BLOCK_SIZE];

    for (int st = start + blockDim.x * 2 * blockIdx.x; st + 2 * blockDim.x < upd_n+1; st += blockDim.x * 2 * gridDim.x) {
        l[threadIdx.x] = arr[st + threadIdx.x];
        __syncthreads();
        r[threadIdx.x] = arr[st + threadIdx.x + blockDim.x];
        __syncthreads();

        if (threadIdx.x == 0) {
            int rc = 0;
            int lc = 0;
            int it = st; //?

            // int count = 0;
            while (true) {
                // count++;
                if (rc == batch)
                    arr[it] = l[lc++];
                else if (lc == batch)
                    arr[it] = r[rc++];
                else {
                    if (l[lc] < r[rc])
                        arr[it] = l[lc++];
                    else if (l[lc] > r[rc])
                        arr[it] = r[rc++];
                    else
                        arr[it] = l[lc++];
                }
                it++;
                if (lc == batch && rc == batch)
                    break;
                // if (count == blockDim.x * 2)
                //     break;
            }
        }
    }

    __syncthreads();
}

int main() {
    bool verbose = true; // 0 for binary, 1 for normal
    int n, upd_n;

    if (verbose)
        cin >> n;
    else
        fread(&n, 4, 1, stdin);

    if (n % BLOCK_SIZE != 0)
        upd_n = (n / BLOCK_SIZE + 1) * BLOCK_SIZE;
    else
        upd_n = n;

    int * arr = (int *)malloc(upd_n * sizeof(int));

    if (verbose)
        for (int i = 0; i < n; i++)
            cin >> arr[i];
    else
        fread(arr, 4, n, stdin);

    for (int i = n; i < upd_n; i++)
        arr[i] = INT_MAX;

    int * ARR_DEV;
    CSC(hipMalloc(&ARR_DEV, sizeof(int) * upd_n));
    CSC(hipMemcpy(ARR_DEV, arr, sizeof(int) * upd_n, hipMemcpyHostToDevice));

    // odd even sort
    for (int i = 0; i < n; i++) {
        oddEvenSortingStep <<<NUM_BLOCKS,BLOCK_SIZE>>> (ARR_DEV, i, n, BLOCK_SIZE);
    }

    // bitonic merge sort
    for (int i = 0; i < 2 * (upd_n / BLOCK_SIZE); i++) {
        if (i % 2 == 0) {
            mergeGPU <<<NUM_BLOCKS,BLOCK_SIZE>>> (ARR_DEV, upd_n, BLOCK_SIZE, 0);
        } else {
            mergeGPU <<<NUM_BLOCKS,BLOCK_SIZE>>> (ARR_DEV, upd_n, BLOCK_SIZE, BLOCK_SIZE);
        }
    }

    CSC(hipGetLastError());
    CSC(hipMemcpy(arr, ARR_DEV, sizeof(int) * upd_n, hipMemcpyDeviceToHost));

    if (verbose) {
        // cout << upd_n << ' ' << n << endl;
        for (int i = 0; i < n; i++) {
            // if (i % BLOCK_SIZE == 0)
                // cout << "| ";
            cout << arr[i] << " ";
        }
        cout << endl;
        // cout << "|" << endl;
    } else {
        fwrite(arr, 4, n, stdout);
    }




    CSC(hipFree(ARR_DEV));
    free(arr);
    return 0;
}
