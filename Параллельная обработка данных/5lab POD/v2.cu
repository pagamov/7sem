#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <sstream>
#include <string>
#include <vector>
// #include <algorithm>
// #include <climits>
#include <thrust/swap.h>
// #include <thrust/extrema.h>
// #include <thrust/functional.h>
// #include <thrust/host_vector.h>
// #include <thrust/device_vector.h>

using namespace std;

#define CSC(call)                                                   \
do {                                                                \
    hipError_t res = call;                                         \
    if (res != hipSuccess) {                                       \
        fprintf(stderr, "ERROR in %s:%d. Message: %s\n",            \
                __FILE__, __LINE__, hipGetErrorString(res));       \
        exit(0);                                                    \
    }                                                               \
} while(0)

#define NUM_BLOCKS 10
#define BLOCK_SIZE 1024

__global__ void oddEvenSortingStep(int * A, int i, int n, int batch) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int shift = blockDim.x * gridDim.x;
    for (int start = idx * batch; start < n; start += shift * batch) {
        for (int j = start + (i % 2); j + 1 < min(start + batch, n); j += 2) {
            if (A[j] > A[j + 1]) {
                thrust::swap(A[j], A[j + 1]);
            }
        }
    }
}

__device__ void swap_step(int* nums, int* tmp, int size, int start, int stop, int step, int i) {
	__shared__ int sh[BLOCK_SIZE];

	for (int shift = start; shift < stop; shift += step) {
		tmp = nums + shift;

		if (i >= BLOCK_SIZE / 2)
			sh[i] = tmp[BLOCK_SIZE * 3 / 2 - 1 - i];
		else
			sh[i] = tmp[i];
		__syncthreads();

		for (int j = BLOCK_SIZE / 2; j > 0; j /= 2) {
			unsigned int XOR = i ^ j;
			// The threads with the lowest ids sort the array
			if (XOR > i) {
				if ((i & BLOCK_SIZE) != 0) {
					if (sh[i] < sh[XOR])
						thrust::swap(sh[i], sh[XOR]);
				} else {
					if (sh[i] > sh[XOR])
						thrust::swap(sh[i], sh[XOR]);
				}
			}
			__syncthreads();
		}

		tmp[i] = sh[i];
	}
}

__global__ void kernel_bitonic_merge_step(int * nums, int size, bool is_odd) {
	// Temporary array for splitting into blocks
	int * tmp = nums;

	// Every thread gets exactly one value in the unsorted array
	unsigned int i = threadIdx.x;
	int id_block = blockIdx.x;
	int offset = gridDim.x;

	// For odd step
	if(is_odd) {
		swap_step(nums, tmp, size, (BLOCK_SIZE / 2) + id_block * BLOCK_SIZE, size - BLOCK_SIZE, offset * BLOCK_SIZE, i);
	} else { // For even step
		swap_step(nums, tmp, size, id_block * BLOCK_SIZE, size, offset * BLOCK_SIZE, i);
	}
}

int main() {
	bool verbose = false; // 0 for binary, 1 for normal
	int n, upd_n;

	if (verbose)
        cin >> n;
    else
        fread(&n, 4, 1, stdin);

	upd_n = ceil((double)n / BLOCK_SIZE) * BLOCK_SIZE;
	int * data = (int *)malloc(sizeof(int) * upd_n);
	int * dev_data;
	CSC(hipMalloc(&dev_data, sizeof(int) * upd_n));

	if (verbose)
        for (int i = 0; i < n; i++)
            cin >> data[i];
    else
        fread(data, 4, n, stdin);

	for (int i = n; i < upd_n; ++i)
		data[i] = INT_MAX;

	CSC(hipMemcpy(dev_data, data, sizeof(int) * upd_n, hipMemcpyHostToDevice));

	for (int i = 0; i < BLOCK_SIZE; i++)
        oddEvenSortingStep <<<NUM_BLOCKS,BLOCK_SIZE>>> (dev_data, i, n, BLOCK_SIZE);

	for (int i = 0; i < 2 * (upd_n / BLOCK_SIZE); i++)
		kernel_bitonic_merge_step<<<NUM_BLOCKS, BLOCK_SIZE>>>(dev_data, upd_n, (bool)(i % 2));

	CSC(hipGetLastError());

	CSC(hipMemcpy(data, dev_data, sizeof(int) * upd_n, hipMemcpyDeviceToHost));

	if (verbose) {
        for (int i = 0; i < n; i++)
            cout << data[i] << " ";
        cout << endl;
    } else {
        fwrite(data, 4, n, stdout);
    }

	CSC(hipFree(dev_data));
	free(data);
	return 0;
}
